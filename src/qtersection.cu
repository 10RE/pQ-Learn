
#include <hip/hip_runtime.h>
#include <iostream>
#include "math.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

using namespace std;

#define SIZE_X 8
#define SIZE_Y 8
#define MESH_SIZE SIZE_X*SIZE_Y
#define ACTION_SIZE 2
#define SURROUNDING_SIZE 9
#define Q_KERNEL_SIZE SURROUNDING_SIZE * MESH_SIZE
#define MAX_VEHICLE_NUM 8

#define LANE_SIZE 4

#define STATE_SIZE (int)pow(MAX_VEHICLE_NUM, LANE_SIZE)
#define Q_TABLE_SIZE STATE_SIZE * ACTION_SIZE
#define TOTAL_Q_TABLE_SIZE Q_TABLE_SIZE * MESH_SIZE

#define ENV_ITEM_SIZE 15

#define ENV_SIZE MESH_SIZE*ENV_ITEM_SIZE

#define BLOCK_SIZE 1024
#define BLOCK_SQUARE_SIZE 32

#define ALPHA 0.5
#define BETA 0.1 //decay reward from surroundings
#define GAMMA 0.1
#define EPSILON 0.4
#define VEHICLE_RATE 0.3

#define VERBOSE 0

#define NUM_TRAIN 1000000
#define NUM_TEST 10000
#define NUM_TEST_ITER 20

#define NO_SURROUNDING 0

#define DEBUG 0

#define SHOW_STATE 0

#define USE_CUDA 0

__global__ void get_coverage_kernel(int* coverage, double* q_table) {
    __shared__ int not_covered_count;
    if (threadIdx.x == 0) {
        not_covered_count = 0;
    }
    int state = threadIdx.x + blockIdx.x * blockDim.x;
    if (state >= (TOTAL_Q_TABLE_SIZE / 2)) {
        return;
    }
    int main_offset = state * ACTION_SIZE;
    bool covered_flag = false;
    for (int i = 0; i < ACTION_SIZE; i++) {
        if (q_table[main_offset + i] != 0) {
            covered_flag = true;
            break;
        }
    }
    if (!covered_flag) {
        atomicAdd(&not_covered_count, 1);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        coverage[blockIdx.x] = not_covered_count;
    }
}

double get_coverage(double* q_table) {
    dim3 block(BLOCK_SIZE);
    int grid_size = TOTAL_Q_TABLE_SIZE / ACTION_SIZE / block.x;
    dim3 grid(grid_size);
    int* cover_count = new int[grid_size];
    int* cover_count_cuda;
    if (hipMalloc(&cover_count_cuda, sizeof(int) * grid_size) != hipSuccess) {
        printf("hipMalloc for coverage failed\n");
    }
    get_coverage_kernel<<<grid, block>>>(cover_count_cuda, q_table);
    hipMemcpy(cover_count, cover_count_cuda, sizeof(int) * grid_size, hipMemcpyDeviceToHost);
    int total = 0;
    //cout << TOTAL_Q_TABLE_SIZE << " " << grid_size << endl;
    for (int i = 0; i < grid_size; i++) {
        //cout << cover_count[i] << " ";
        total += cover_count[i];
    }
    //cout << total << endl;
    hipFree(cover_count_cuda);
    return 1 - (double)total / (double)(TOTAL_Q_TABLE_SIZE / ACTION_SIZE);
}

/*
0: cur_state_up
1: cur_state_down
2: cur_state_left
3: cur_state_right
4: cur_state_id
5: next_state_up
6: next_state_down
7: next_state_left
8: next_state_right
9: next_state_id
10: action // not used
11: in_vehicle_up
12: in_vehicle_down
13: in_vehicle_left
14: in_vehicle_right
*/
//action
__device__ int get_env(int* env, int mesh_id, int item_id) {
    return env[mesh_id * ENV_ITEM_SIZE + item_id];
}

__device__ void set_env(int* env, int mesh_id, int item_id, double value) {
    env[mesh_id * ENV_ITEM_SIZE + item_id] = value;
}

__device__ void set_env(int* env, int id, double value) {
    env[id] = value;
}


__device__ int* get_cur_state(int* env, int mesh_id) {
    return env + mesh_id * ENV_ITEM_SIZE;
}

__device__ int get_cur_state_id(int* env, int mesh_id) {
    return *(env + mesh_id * ENV_ITEM_SIZE + 4);
}

__device__ void set_cur_state_id(int* env, int mesh_id, int cur_state_id) {
    *(env + mesh_id * ENV_ITEM_SIZE + 4) = cur_state_id;
}

__device__ int* get_next_state(int* env, int mesh_id) {
    return env + mesh_id * ENV_ITEM_SIZE + 5;
}

__device__ void set_next_state_id(int* env, int mesh_id, int next_state_id) {
    *(env + mesh_id * ENV_ITEM_SIZE + 9) = next_state_id;
}

__device__ int get_next_state_id(int* env, int mesh_id) {
    return *(env + mesh_id * ENV_ITEM_SIZE + 9);
}

// __device__ double* get_reward(int* env, int id) {
//     return env + id * ENV_ITEM_SIZE + 8;
// }

__device__ int get_action(int* env, int mesh_id) {
    return *(env + mesh_id * ENV_ITEM_SIZE + 10);
}

__device__ void set_action(int* env, int mesh_id, int action) {
    *(env + mesh_id * ENV_ITEM_SIZE + 10) = action;
}

__device__ int* get_in_vehicle(int* env, int mesh_id) {
    return env + mesh_id * ENV_ITEM_SIZE + 11;
}

__device__ void increase_in_vehicle(int* env, int mesh_id, int lane) {
    *(env + mesh_id * ENV_ITEM_SIZE + 11 + lane) += 1;
}

__global__ void reset_env_kernel(int* env, bool* is_end_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= ENV_SIZE) {
        return;
    }
    if (*is_end_state) {
        set_env(env, id, 0);
        return;
    }
    
}

__device__ int get_state(int* state) {
    int state_ret = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        state_ret += state[i] * (int)pow(MAX_VEHICLE_NUM, i);
    }
    return state_ret;
}

__device__ int get_next_state_cuda(int* next_state, int* state, int action){
    int lane = action * 2;
    for (int i = 0; i < LANE_SIZE; i++){
        if (i == lane || i == (lane + 1)) {
            next_state[i] = state[i] > 0 ? state[i] - 1 : state[i];
        }
        else {
            next_state[i] = state[i];
        }            
    }
    return get_state(next_state);
}

__device__ double get_value_cuda(double* q_table, int id, int state, int action){
    return q_table[Q_TABLE_SIZE * id + state * ACTION_SIZE + action];
}

__device__ void set_value_cuda(double* q_table, int id, int state, int action, double value){
    q_table[Q_TABLE_SIZE * id + state * ACTION_SIZE + action] = value;
}

// __global__ void get_max_q_value_in_action(double* max_q, double* isMAX, double* q_table, int state) {
//     int row = threadIdx.x + blockIdx.x * blockDim.x;
//     int col = threadIdx.y + blockIdx.y * blockDim.y;
//     isMAX[row] = true;
//     if (get_value_cuda(q_table, state, row) < get_value_cuda(q_table, state, col)) {
//         isMAX[row] = false;
//     }
//     if (isMAX[row]) {
//         *max_q = get_value_cuda(q_table, state, row);
//     }
// }

__device__ double get_max_q_value_in_action_cuda(double* q_table, int id, int state) {
    double max_q = -999;
    for (int i = 0; i < ACTION_SIZE; i++) {
        double value = get_value_cuda(q_table, id, state, i);
        
        if (value > max_q) {
            max_q = value;
        }
    }
    //printf("max_value: %f\n", max_q);
    return max_q;
}


/* can be a global */
__device__ double cal_reward_cuda(int* next_state, int* state, int* in_vehicle) {
    int max_next_lane_num = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        if (next_state[i] > max_next_lane_num) {
            max_next_lane_num = next_state[i];
        }
    }
    int max_cur_lane_num = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        if (state[i] > max_cur_lane_num) {
            max_cur_lane_num = state[i];
        }
    }
    double in_vehicle_num = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        in_vehicle_num += in_vehicle[i];
    }
    double reward = max_cur_lane_num * max_cur_lane_num - max_next_lane_num * max_next_lane_num - in_vehicle_num * 0.025;

    return reward;
}

__device__ int choose_max_action_cuda(int mesh_id, double* q_table, int* env) {//, double* max_q) {
    int state_id = get_cur_state_id(env, mesh_id);
    double max_q = -999;
    int action_id = 0;
    #if DEBUG
        double value0 = get_value_cuda(q_table, mesh_id, state_id, 0);
        double value1 = get_value_cuda(q_table, mesh_id, state_id, 1);
    #endif

    for (int i = 0; i < ACTION_SIZE; i++) {
        double value = get_value_cuda(q_table, mesh_id, state_id, i);
        
        if (value > max_q) {
            max_q = value;
            action_id = i;
        }
    }
    #if DEBUG
    printf("max_action %d, %d, %d: %f, %f\n", mesh_id, action_id, state_id, value0, value1);
    #endif
    //printf("max_value: %f\n", max_q);
    return action_id;
}

__device__ int choose_random_action_cuda(int mesh_id, hiprandState* rand_state) {
    double rand_num = hiprand_uniform(&rand_state[mesh_id]);
    return (int) (rand_num * ACTION_SIZE);
}

__device__ int choose_action_cuda(int mesh_id, double* q_table, int* env, hiprandState* rand_state) {
    double rand_num = hiprand_uniform(&rand_state[mesh_id]);
    if (rand_num < EPSILON) {
        return choose_random_action_cuda(mesh_id, rand_state);
    }
    else {
        return choose_max_action_cuda(mesh_id, q_table, env);
    }
}

__device__ void cal_vehicle_out (int* environment, int mesh_id, int action) {
    if (action == 0) {
        if (mesh_id - SIZE_X >= 0) {
            //up out vehicle
            set_env(environment, mesh_id - SIZE_X, 12, get_cur_state(environment, mesh_id)[1] > 0 ? 1 : 0);
        }
        if (mesh_id + SIZE_X < MESH_SIZE) {
            //down out vehicle
            set_env(environment, mesh_id + SIZE_X, 11, get_cur_state(environment, mesh_id)[0] > 0 ? 1 : 0);
        }
    }
    else {
        if (mesh_id % SIZE_X != 0) {
            //left out vehicle
            set_env(environment, mesh_id - 1, 14, get_cur_state(environment, mesh_id)[3] > 0 ? 1 : 0);
        }
        if (mesh_id % SIZE_X != SIZE_X - 1) {
            //down out vehicle
            set_env(environment, mesh_id + 1, 13, get_cur_state(environment, mesh_id)[2] > 0 ? 1 : 0);
        }
    }
    #if DEBUG
        __syncthreads();
        printf("out_vehicle: %d, %d, %d, %d, %d\n", mesh_id, get_in_vehicle(environment, mesh_id)[0], get_in_vehicle(environment, mesh_id)[1], get_in_vehicle(environment, mesh_id)[2], get_in_vehicle(environment, mesh_id)[3]);
    #endif
}

__device__ void update_vehicle_in_with_out (int* environment, int mesh_id, int action) {
    if (action == 0) {
        set_env(environment, mesh_id, 11, get_in_vehicle(environment, mesh_id)[2]);
        set_env(environment, mesh_id, 12, get_in_vehicle(environment, mesh_id)[3]);
        set_env(environment, mesh_id, 13, 0);
        set_env(environment, mesh_id, 14, 0);
    }
    else {
        set_env(environment, mesh_id, 13, get_in_vehicle(environment, mesh_id)[0]);
        set_env(environment, mesh_id, 14, get_in_vehicle(environment, mesh_id)[1]);
        set_env(environment, mesh_id, 11, 0);
        set_env(environment, mesh_id, 12, 0);
    }
}

__device__ void update_vehicle_in (int* environment, int mesh_id) {
    for (int i = 0; i < LANE_SIZE; i++) {
        set_env(environment, mesh_id, 5 + i, get_next_state(environment, mesh_id)[i] + get_in_vehicle(environment, mesh_id)[i]);
        //set_env(environment, mesh_id, 11 + i, 0);
    }
    #if DEBUG
        printf("in_vehicle: %d, %d, %d, %d, %d\n", mesh_id, get_in_vehicle(environment, mesh_id)[0], get_in_vehicle(environment, mesh_id)[1], get_in_vehicle(environment, mesh_id)[2], get_in_vehicle(environment, mesh_id)[3]);
        int* state = get_cur_state(environment, mesh_id);
        int* next_state = get_next_state(environment, mesh_id);
        printf("update__vehicle_in: %d, cur_state: %d, %d, %d, %d, next_state: %d, %d, %d, %d \n", mesh_id, state[0], state[1], state[2], state[3], next_state[0], next_state[1], next_state[2], next_state[3]);
    #endif
}

__device__ void reset_vehicle_in (int* environment, int mesh_id) {
    for (int i = 0; i < LANE_SIZE; i++) {
        set_env(environment, mesh_id, 11 + i, 0);
    }
}

__global__ void update_env_pre_kernel(int* environment, double* q_table, hiprandState* rand_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= MESH_SIZE) {
        return;
    }
    int* state = get_cur_state(environment, id);
    int cur_state_id = get_state(state);
    set_cur_state_id(environment, id, cur_state_id);

    int action = choose_action_cuda(id, q_table, environment, rand_state);

    set_action(environment, id, action);
    
    int next_state_id = get_next_state_cuda(get_next_state(environment, id), state, action);  // update next state, get next state id
    #if SHOW_STATE
        printf("env_prev: %d, action: %d, cur_state: %d, %d, %d, %d, next_state: %d, %d, %d, %d \n", id, action, state[0], state[1], state[2], state[3], get_next_state(environment, id)[0], get_next_state(environment, id)[1], get_next_state(environment, id)[2], get_next_state(environment, id)[3]);
    #endif
    reset_vehicle_in(environment, id);
    set_next_state_id(environment, id, next_state_id);
    //__syncthreads();
    cal_vehicle_out(environment, id, action);
    //__syncthreads();
    //update_vehicle_in_with_out(environment, id, action);
}

__global__ void update_env_pre_run_kernel(int* environment, double* q_table, hiprandState* rand_state, bool show_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= MESH_SIZE) {
        return;
    }
    int* state = get_cur_state(environment, id);
    int cur_state_id = get_state(state);
    set_cur_state_id(environment, id, cur_state_id);

    int action = choose_max_action_cuda(id, q_table, environment);
    
    set_action(environment, id, action);

    int next_state_id = get_next_state_cuda(get_next_state(environment, id), state, action);  // update next state, get next state id
    if (show_state) {
        printf("env_prev: %d, action: %d, cur_state: %d, %d, %d, %d, next_state: %d, %d, %d, %d, q_0: %f, q_1: %f \n", id, action, state[0], state[1], state[2], state[3], get_next_state(environment, id)[0], get_next_state(environment, id)[1], get_next_state(environment, id)[2], get_next_state(environment, id)[3], get_value_cuda(q_table, id, get_cur_state_id(environment, id), 0), get_value_cuda(q_table, id, get_cur_state_id(environment, id), 1));
    }
    reset_vehicle_in(environment, id);
    set_next_state_id(environment, id, next_state_id);
    //__syncthreads();
    cal_vehicle_out(environment, id, action);
    //__syncthreads();
    //update_vehicle_in_with_out(environment, id, action);
}

__global__ void update_reward_kernel(int* environment, double* reward) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int mesh_id = x;
    if (mesh_id >= MESH_SIZE) {
        return;
    }

    int max_next_lane_num = 0;
    int next_lane_sum = 0;
    int* next_state = get_next_state(environment, mesh_id);
    // if (action_id == get_action(environment, mesh_id)) {
    //     next_state = get_next_state(environment, mesh_id);
    // } else {
    //     next_state = ;
    // }
    for (int i = 0; i < LANE_SIZE; i++){
        
        next_lane_sum += next_state[i];
        if (next_state[i] > max_next_lane_num) {
            max_next_lane_num = next_state[i];
        }
    }
    int max_cur_lane_num = 0;
    int cur_lane_sum = 0;
    int* cur_state = get_cur_state(environment, mesh_id);
    for (int i = 0; i < LANE_SIZE; i++){
        cur_lane_sum += cur_state[i];
        if (cur_state[i] > max_cur_lane_num) {
            max_cur_lane_num = cur_state[i];
        }
    }

    double in_vehicle_num = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        in_vehicle_num += get_in_vehicle(environment, mesh_id)[i];
    }
    reward[mesh_id] = 0.25 * (cur_lane_sum - next_lane_sum) + max_cur_lane_num * max_cur_lane_num - max_next_lane_num * max_next_lane_num - in_vehicle_num * 0.025;
    //reward[mesh_id] = max_cur_lane_num * max_cur_lane_num - max_next_lane_num * max_next_lane_num - in_vehicle_num * 0.025;
    #if DEBUG
        printf("reward: %d, %f\n", mesh_id, reward[mesh_id]);
    #endif
}

__device__ double cal_reward(int* environment, int mesh_id, int* cur_state, int* next_state) {
    int max_next_lane_num = 0;
    int next_lane_sum = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        next_lane_sum += next_state[i];
        if (next_state[i] > max_next_lane_num) {
            max_next_lane_num = next_state[i];
        }
    }
    int max_cur_lane_num = 0;
    int cur_lane_sum = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        cur_lane_sum += cur_state[i];
        if (cur_state[i] > max_cur_lane_num) {
            max_cur_lane_num = cur_state[i];
        }
    }

    double in_vehicle_num = 0;
    for (int i = 0; i < LANE_SIZE; i++){
        in_vehicle_num += get_in_vehicle(environment, mesh_id)[i];
    }
    return 0.25 * (cur_lane_sum - next_lane_sum) + max_cur_lane_num * max_cur_lane_num - max_next_lane_num * max_next_lane_num - in_vehicle_num * 0.025;
}

__global__ void debug_reward_kernel(double* reward) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int mesh_id = x + y * blockDim.x * gridDim.x;
    if (mesh_id >= MESH_SIZE) {
        return;
    }
    printf("reward_debug: %d, %f\n", mesh_id, reward[mesh_id]);
}

__global__ void cal_q_kernel_single(int* environment, double* reward, double* q_table) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int action = threadIdx.y;

    int surrounding_id = id % SURROUNDING_SIZE;
    #if NO_SURROUNDING
    if (surrounding_id != 0) {
        return;
    }
    #endif
    int mesh_id = id / SURROUNDING_SIZE;
    #if NO_SURROUNDING 
    if (action != get_action(environment, mesh_id)) {
        return;
    }
    #endif
    if (id >= Q_KERNEL_SIZE) {
        return;
    }
    if (action >= ACTION_SIZE) {
        return;
    }
    
    
    int* state = get_cur_state(environment, mesh_id);
    int actual_state[LANE_SIZE] = {0, 0, 0, 0};//new int[LANE_SIZE];
    int next_state[LANE_SIZE] = {0, 0, 0, 0};
    if (surrounding_id == 0) {
        for (int i = 0; i < LANE_SIZE; i++){
            actual_state[i] = (int)state[i];
        }
    }
    else {
        for (int i = 0; i < LANE_SIZE; i++){
            if ((surrounding_id - 1) / 2 == i) {
                int tmp_state = state[i] + (surrounding_id % 2 == 0 ? 1 : -1);
                if (tmp_state < 0 || tmp_state > MAX_VEHICLE_NUM) {
                    return;
                }
                actual_state[i] = tmp_state > 0 ? (tmp_state < MAX_VEHICLE_NUM ? tmp_state : MAX_VEHICLE_NUM) : 0;
            }
            else {
                actual_state[i] = (int)state[i];
            }
        }
    }

    #if NO_SURROUNDING
        int state_int = get_state(actual_state);
        int state_int = get_cur_state_id(environment, mesh_id);

        double cur_q = get_value_cuda(q_table, mesh_id, state_int, action);
        int next_state_int = get_next_state_id(environment, mesh_id);

        double local_reward = reward[mesh_id];

    #else
        int state_int = get_state(actual_state);
    
        double cur_q = get_value_cuda(q_table, mesh_id, state_int, action);
        int next_state_int = get_next_state_cuda(next_state, actual_state, action);

        double local_reward = cal_reward(environment, mesh_id, actual_state, next_state);
    #endif 

    double next_q = 0;
    next_q = get_max_q_value_in_action_cuda(q_table, mesh_id, next_state_int);

    //get_max_q_value_in_action<<<ACTION_SIZE, ACTION_SIZE>>>(&max_q, isMAX, q_table, next_state);
    //cudaFree(isMAX);
    //double local_reward = reward[mesh_id];
    
    double surrounding_reward = 0;

    int s_id = mesh_id - SIZE_X;
    if (s_id > 0) {
        surrounding_reward += reward[s_id];
    }
    s_id = mesh_id + SIZE_X;
    if (s_id < MESH_SIZE) {
        surrounding_reward += reward[s_id];
    }
    s_id = mesh_id - 1;
    if (mesh_id % SIZE_X != 0) {
        surrounding_reward += reward[s_id];
    }
    s_id = mesh_id + 1;
    if (mesh_id % SIZE_X != (SIZE_X - 1)) {
        surrounding_reward += reward[s_id];
    }

    double q_value = 0;
    q_value += (1 - ALPHA) * cur_q;
    q_value += ALPHA * local_reward + BETA * surrounding_reward;
    q_value += ALPHA * GAMMA * next_q;
    //__syncthreads();
    set_value_cuda(q_table, mesh_id, state_int, action, q_value);

    #if DEBUG
    __syncthreads();
    printf("q_value %d state: %d, %d, %d, %d, %d, action: %d, next_state: %d, %d, %d, %d, q_val: %f, reward: %f, surrounding: %f, q_val_0: %f, q_val_1: %f\n", mesh_id, state_int, actual_state[0], actual_state[1], actual_state[2], actual_state[3], action, next_state[0], next_state[1], next_state[2], next_state[3], next_q, local_reward, surrounding_reward, get_value_cuda(q_table, mesh_id, state_int, 0), get_value_cuda(q_table, mesh_id, state_int, 1));
    #endif
}

__device__ void generate_new_vehicel(int* environment, int mesh_id, hiprandState* rand_state) {
    if (mesh_id < SIZE_X) {
        double rand_num = hiprand_uniform(&rand_state[mesh_id]);
        if (rand_num < VEHICLE_RATE) {
            increase_in_vehicle(environment, mesh_id, 0);
        }
    }
    if (mesh_id >= MESH_SIZE - SIZE_X) {
        double rand_num = hiprand_uniform(&rand_state[mesh_id]);
        if (rand_num < VEHICLE_RATE) {
            increase_in_vehicle(environment, mesh_id, 1);
        }
    }
    if (mesh_id % SIZE_X == 0) {
        double rand_num = hiprand_uniform(&rand_state[mesh_id]);
        if (rand_num < VEHICLE_RATE) {
            increase_in_vehicle(environment, mesh_id, 2);
        }
    }
    if (mesh_id % SIZE_X == SIZE_X - 1) {
        double rand_num = hiprand_uniform(&rand_state[mesh_id]);
        if (rand_num < VEHICLE_RATE) {
            increase_in_vehicle(environment, mesh_id, 3);
        }
    }
}

__device__ void take_next_step(int* environment, int mesh_id) {
    for (int i = 0; i < LANE_SIZE; i++) {
        set_env(environment, mesh_id, i, get_next_state(environment, mesh_id)[i]);
    }
    //set_env(environment, mesh_id, 4, get_next_state_id(environment, mesh_id));
}

__global__ void update_env_after_kernel(int* environment, double* q_table, hiprandState* rand_state) {
    int mesh_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (mesh_id >= MESH_SIZE) {
        return;
    }
    generate_new_vehicel(environment, mesh_id, rand_state);
    update_vehicle_in(environment, mesh_id);
    take_next_step(environment, mesh_id);
}

__global__ void is_end_state_kernel(int* environment, bool* is_end) {
    int mesh_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (mesh_id >= MESH_SIZE) {
        return;
    }
    int* state = get_cur_state(environment, mesh_id);
    for (int i = 0; i < LANE_SIZE; i++) {
        if (state[i] > MAX_VEHICLE_NUM) {
            *is_end = true;
            #if DEBUG
                printf("Is end state: %d state: %d %d %d %d\n", mesh_id, state[0], state[1], state[2], state[3]);
            #endif
            return;
        }
    }
    #if DEBUG
        printf("Is not end state: %d state: %d %d %d %d\n", mesh_id, state[0], state[1], state[2], state[3]);
    #endif
    return;
}

__global__ void reset_is_end_state_kernel(bool* is_end) {
    *is_end = false;
}

// __global__ void get_coverage_kernel(int* coverage, double* q_table) {
//     __shared__ int not_covered_count;
//     if (threadIdx.x == 0) {
//         not_covered_count = 0;
//     }
//     int state = threadIdx.x + blockIdx.x * blockDim.x;
//     if (state >= STATE_SIZE) {
//         return;
//     }
//     int main_offset = state * ACTION_SIZE;
//     bool covered_flag = false;
//     for (int i = 0; i < ACTION_SIZE; i++) {
//         if (get_value_cuda(q_table,  state, i) != 0) {
//             covered_flag = true;
//             break;
//         }
//     }
//     if (!covered_flag) {
//         atomicAdd(&not_covered_count, 1);
//     }
//     __syncthreads();
//     if (threadIdx.x == 0) {
//         coverage[blockIdx.x] = not_covered_count;
//     }
// }

void malloc_q_table(double* q_table, int size) {
    if(hipMalloc(&q_table, sizeof(double) * size) != hipSuccess){
        cout << "Could not allocate q_table on GPU" << endl;
    }
}

__global__ void rand_setup_kernel(hiprandState* rand_state)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), tid, 0, &rand_state[tid]);
}

void train(double* qtable) {
    int train_step = 0;
    int epoch_step = 0;

    hiprandState* rand_states;
    if (hipMalloc(&rand_states, sizeof(hiprandState) * MESH_SIZE) != hipSuccess) {
        cout << "hiprandState malloc failed" << endl;
    }
    rand_setup_kernel<<<MESH_SIZE, 1>>>(rand_states);

    // cout << train_step << endl;
    //QTable qtable = QTable();

    int* environment;
    if (hipMalloc(&environment, sizeof(int) * ENV_SIZE) != hipSuccess) {
        cout << "env malloc failed" << endl;
    }
    double* reward;
    if (hipMalloc(&reward, sizeof(double) * MESH_SIZE * ACTION_SIZE) != hipSuccess) {
        cout << "reward malloc failed" << endl;
    }
    bool* is_end_state_cuda;
    bool is_end_state = false;
    if (hipMalloc(&is_end_state_cuda, sizeof(bool)) != hipSuccess) {
        cout << "is_end_state malloc failed" << endl;
    }
    int* environment_cpu = (int*)malloc(sizeof(int) * ENV_SIZE);
    dim3 env_block(ENV_SIZE < BLOCK_SIZE ? (ENV_SIZE > 32 ? ENV_SIZE : 32) : BLOCK_SIZE);
    dim3 env_grid(ENV_SIZE / env_block.x + 1);
    dim3 mesh_block(MESH_SIZE < BLOCK_SQUARE_SIZE ? MESH_SIZE : BLOCK_SQUARE_SIZE);
    dim3 mesh_grid(MESH_SIZE / mesh_block.x + 1);
    dim3 reward_block(MESH_SIZE < BLOCK_SIZE ? MESH_SIZE : BLOCK_SIZE);
    dim3 reward_grid(MESH_SIZE / mesh_block.x + 1);
    dim3 q_block(Q_KERNEL_SIZE < BLOCK_SIZE / ACTION_SIZE ? Q_KERNEL_SIZE : BLOCK_SIZE / ACTION_SIZE, ACTION_SIZE);
    dim3 q_grid(Q_KERNEL_SIZE / q_block.x + 1);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    while (train_step < NUM_TRAIN) {
        // cout << "======================" << endl;
        // cout << "start epoch: " << epoch_step << endl;
            
        #if DEBUG
            // if (cudaMemcpy(
            //     environment_cpu, 
            //     environment, 
            //     sizeof(int) * ENV_SIZE,
            //     cudaMemcpyDeviceToHost
            // ) != cudaSuccess) {
            //     cout << "Could not copy to CPU" << endl;
            // }
            // for (int i = 0; i < MESH_SIZE; i++) {
            //     cout << "intersection: " << i << endl;
            //     for (int j = 0; j < LANE_SIZE; j++) {
            //         cout << environment_cpu[i * ENV_ITEM_SIZE + j] << " ";
            //     }
            //     cout << endl;
            // }
        #endif
        
        is_end_state_kernel<<<mesh_grid, mesh_block>>>(environment, is_end_state_cuda);
        //cudaEventRecord(start, 0);
        reset_env_kernel<<<env_grid, env_block>>>(environment, is_end_state_cuda);
        reset_is_end_state_kernel<<<1, 1>>>(is_end_state_cuda);

        // cout << "----------------------" << endl;
        // cout << "train step: " << train_step << endl;
        
        //debug_reward_kernel<<<mesh_grid, mesh_block>>>(reward);
        
        update_env_pre_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states);
        update_reward_kernel<<<reward_grid, reward_block>>>(environment, reward);
        cal_q_kernel_single<<<q_grid, q_block>>>(environment, reward, qtable);
        update_env_after_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states);
        
        
        //cudaDeviceSynchronize();
        train_step ++;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "Time: " << time << endl;

    hipFree(environment);
    hipFree(reward);
    hipFree(is_end_state_cuda);
    hipFree(rand_states);

    // double* qtable_cpu = (double*)malloc(sizeof(double) * MESH_SIZE * q_table_size);
    // if (cudaMemcpy(
    //     qtable_cpu, 
    //     qtable, 
    //     sizeof(double) * MESH_SIZE * q_table_size,
    //     cudaMemcpyDeviceToHost
    // ) != cudaSuccess){
    //     cout << "Could not copy to CPU" << endl;
    // }
    // for (int i = 0; i < MESH_SIZE; i++) {
    //     cout << "mesh: " << i << endl;
    //     for (int j = 0; j < state_size; j++) {
    //         cout << "state: " << j << endl;
    //         for (int k = 0; k < ACTION_SIZE; k++) {
    //             cout << qtable_cpu[i * q_table_size + j * ACTION_SIZE + k] << " ";
    //         }
    //         cout << endl;
    //     }
    // }

}

int run(double* qtable) {
    int run_step = 0;

    hiprandState* rand_states;
    if (hipMalloc(&rand_states, sizeof(hiprandState) * MESH_SIZE) != hipSuccess) {
        cout << "hiprandState malloc failed" << endl;
    }
    rand_setup_kernel<<<MESH_SIZE, 1>>>(rand_states);
    
    //cout << run_step << endl;

    int* environment;
    if (hipMalloc(&environment, sizeof(int) * ENV_SIZE) != hipSuccess) {
        cout << "env malloc failed" << endl;
    }
    double* reward;
    if (hipMalloc(&reward, sizeof(double) * MESH_SIZE) != hipSuccess) {
        cout << "reward malloc failed" << endl;
    }
    bool* is_end_state_cuda;
    bool is_end_state = false;
    if (hipMalloc(&is_end_state_cuda, sizeof(bool)) != hipSuccess) {
        cout << "is_end_state malloc failed" << endl;
    }
    int* environment_cpu = (int*)malloc(sizeof(int) * ENV_SIZE);
    dim3 env_block(ENV_SIZE < BLOCK_SIZE ? (ENV_SIZE > BLOCK_SQUARE_SIZE ? ENV_SIZE : BLOCK_SQUARE_SIZE) : BLOCK_SIZE);
    dim3 env_grid(ENV_SIZE / env_block.x + 1);
    dim3 mesh_block(MESH_SIZE < BLOCK_SIZE ? MESH_SIZE : BLOCK_SIZE);
    dim3 mesh_grid(MESH_SIZE / mesh_block.x + 1);
    dim3 q_block(SURROUNDING_SIZE < BLOCK_SIZE / ACTION_SIZE ? SURROUNDING_SIZE : BLOCK_SIZE / ACTION_SIZE, ACTION_SIZE);
    dim3 q_grid(SURROUNDING_SIZE / q_block.x + 1);
    reset_is_end_state_kernel<<<1, 1>>>(is_end_state_cuda);
    reset_env_kernel<<<env_grid, env_block>>>(environment, is_end_state_cuda);
    /*
    while (run_step < NUM_TEST) {
        // cout << "======================" << endl;
        // cout << "start epoch: " << epoch_step << endl;
            
        #if DEBUG
            // if (cudaMemcpy(
            //     environment_cpu, 
            //     environment, 
            //     sizeof(int) * ENV_SIZE,
            //     cudaMemcpyDeviceToHost
            // ) != cudaSuccess) {
            //     cout << "Could not copy to CPU" << endl;
            // }
            // for (int i = 0; i < MESH_SIZE; i++) {
            //     cout << "intersection: " << i << endl;
            //     for (int j = 0; j < LANE_SIZE; j++) {
            //         cout << environment_cpu[i * ENV_ITEM_SIZE + j] << " ";
            //     }
            //     cout << endl;
            // }
        #endif
        
        is_end_state_kernel<<<mesh_grid, mesh_block>>>(environment, is_end_state_cuda);
        //cudaEventRecord(start, 0);
        
        if(cudaMemcpy(
            &is_end_state, 
            is_end_state_cuda, 
            sizeof(bool),
            cudaMemcpyDeviceToHost
        ) != cudaSuccess){
            cout << "Could not copy to CPU" << endl;
        }
        if (is_end_state) {
            break;
        }
        else if (run_step > NUM_TEST) {
            break;
        }

        update_env_pre_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states);
        update_env_pre_run_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states, false);
        update_env_after_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states);
        run_step ++;
    }
    */
    
    reset_is_end_state_kernel<<<1, 1>>>(is_end_state_cuda);
    reset_env_kernel<<<env_grid, env_block>>>(environment, is_end_state_cuda);
    while (1) {

        #if DEBUG
            // if (cudaMemcpy(
            //     environment_cpu, 
            //     environment, 
            //     sizeof(int) * ENV_SIZE,
            //     cudaMemcpyDeviceToHost
            // ) != cudaSuccess) {
            //     cout << "Could not copy to CPU" << endl;
            // }
            // for (int i = 0; i < MESH_SIZE; i++) {
            //     cout << "intersection: " << i << endl;
            //     for (int j = 0; j < LANE_SIZE; j++) {
            //         cout << environment_cpu[i * ENV_ITEM_SIZE + j] << " ";
            //     }
            //     cout << endl;
            // }
        #endif

        is_end_state_kernel<<<mesh_grid, mesh_block>>>(environment, is_end_state_cuda);
        hipDeviceSynchronize();
        if(hipMemcpy(
            &is_end_state, 
            is_end_state_cuda, 
            sizeof(bool),
            hipMemcpyDeviceToHost
        ) != hipSuccess){
            cout << "Could not copy to CPU" << endl;
        }
        if (is_end_state) {
            // cout << "test failed at step: " << run_step << endl;
            //reset_env_kernel<<<env_grid, env_block>>>(environment);
            //reset_is_end_state_kernel<<<1, 1>>>(is_end_state_cuda);
            break;
        }
        else if (run_step > NUM_TEST) {
            // cout << "test success after step: " << run_step << endl;
            break;
        }
        reset_is_end_state_kernel<<<1, 1>>>(is_end_state_cuda);
        //cout << "----------------------" << endl;
        //cout << "run step: " << run_step << endl;
        update_env_pre_run_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states, false);
        update_env_after_kernel<<<mesh_grid, mesh_block>>>(environment, qtable, rand_states);
        hipDeviceSynchronize();
        run_step ++;
    }
    hipFree(environment);
    hipFree(reward);
    hipFree(is_end_state_cuda);
    hipFree(rand_states);
    return run_step;

}

int main() {

    // IntersectionMesh mesh = IntersectionMesh();
    // mesh.train();
    // //mesh.print();
    // mesh.run();

    double* qtable;
    int state_size = ((int)pow(MAX_VEHICLE_NUM, LANE_SIZE));
    int q_table_size = state_size * ACTION_SIZE;
    if (hipMalloc(&qtable, sizeof(double) * MESH_SIZE * q_table_size) != hipSuccess) {
        cout << "qtable malloc failed" << endl;
    }

    train(qtable);
    
    cout << "Coverage: " << get_coverage(qtable) << endl;
    int avg_train = 0;
    int success_count = 0;
    for (int i = 0; i < NUM_TEST_ITER; i++) {
        int run_step = run(qtable);
        if (run_step == NUM_TEST + 1) {
            success_count ++;
        }
        avg_train += run_step;
    }
    cout << "Success rate: " << success_count << endl;
    cout << "Average steps: " << avg_train / NUM_TEST_ITER << endl;
    hipFree(qtable);
    return 0;
}
